#include "hip/hip_runtime.h"
#include "bonsai.h"
// #include "support_kernels.cu"
#include "../profiling/bonsai_timing.h"
PROF_MODULE(parallel);

#include <stdio.h>
#include "node_specs.h"

#include <cstdlib>
#include <iostream>
#include <map>
#include <cassert>
#include <algorithm>


#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/pair.h>
#include <thrust/partition.h>
#include <thrust/version.h>

#if THRUST_VERSION >=  100700


#include <thrust/system/cuda/vector.h>
#include <thrust/system/cuda/execution_policy.h>

//Thrust cached allocator, note this assumes that the passed buffer has sufficient
//size for the requested operations!!!!!!!!!!!!!!
// cached_allocator: a simple allocator for caching allocation requests
class cached_allocator
{

private:
  int memOffset;
  int *currentPointer;

  public:

  typedef char value_type;

    cached_allocator(my_dev::dev_mem<uint> &temporaryBuffer, int startOffset)
    {
      memOffset      = 0;
      currentPointer = (int*)temporaryBuffer.a(startOffset);
    }

    ~cached_allocator(){}
    ///////////
    //Return the number of elements (of type uint) to be padded
    //to get to the correct address boundary
     int getGlobalMemAllignmentPadding2(int n)
    {
      const int allignBoundary = 128*sizeof(uint); //CC 2.X and 3.X ,128 bytes

      int offset = 0;
      //Compute the number of bytes
      offset = n*sizeof(uint);
      //Compute number of allignBoundary byte blocks
      offset = (offset / allignBoundary) + (((offset % allignBoundary) > 0) ? 1 : 0);
      //Compute the number of bytes padded / offset
      offset = (offset * allignBoundary) - n*sizeof(uint);
      //Back to the actual number of elements
      offset = offset / sizeof(uint);

      return offset;
    }


    char *allocate(std::ptrdiff_t num_bytes)
    {
      char *result = (char*)(void*)(size_t)(currentPointer + memOffset);

      //Convert num_bytes to integer offset
      int numIntItems = (int)(num_bytes / sizeof(int));
      numIntItems++;

      //Increase the offset, make sure it is a multiple of predefined number
      int currentOffset = memOffset + numIntItems;
      int padding       = getGlobalMemAllignmentPadding2(currentOffset);
      memOffset         = currentOffset + padding;

//      std::cout << "Allocating: bytes: " << num_bytes << std::endl;
//      std::cout << "Allocating: ints : " << numIntItems<< std::endl;
//      std::cout << "memOffset: " <<  memOffset  << std::endl;
//      std::cout << "currentOffset: " << currentOffset<< std::endl;
//      std::cout << "padding: " << padding << std::endl;

      return result;
    }

    void deallocate(char *ptr, size_t n) {}
};

#endif


//Checks the highest bit to see if a particle is in our domain
struct isInOurDomain
{
  __host__ __device__
  bool operator()(const uint2 &val)
  {
    return (val.x >> 31);
  }
};

//Compare the x component to determine if it's within our domain
struct domainCompare
{
  __host__ __device__
  bool operator()(uint2 x, uint2 y)
  {
    return x.x < y.x;
  }
};


 struct domainCompare2 : public binary_function<uint2,uint2,bool>
{
__host__ __device__ bool operator()(const uint2 &lhs, const uint2 &rhs) const {return lhs.x == rhs.x;}
}; // end domainCompare2

#include <sys/time.h>
 double get_time() {

   struct timeval Tvalue;
   struct timezone dummy;

   gettimeofday(&Tvalue,&dummy);
   return ((double) Tvalue.tv_sec +1.e-6*((double) Tvalue.tv_usec));
 }

#if 0
 thrust_partitionDomains
  First step, partition?
  IDs:     [0,1,2,4,5,6,7,3, 8  ,9  ]
  Domains: [0,1,3,1,1,0,3,0xF,0xF,0xF]

  Second step, sort by exported domain
   IDs:     [0,6,1,4,5,3,7,3, 8  ,9  ]
   Domains: [0,0,1,1,1,3,3,0xF,0xF,0xF]

 Third step, reduce the domains
   Domains/Key  [0,0,1,1,1,3,3]
   Values       [1,1,1,1,1,1,1]
   reducebykey  [0,1,3] domain IDs
                [2,3,2] # particles per domain
#endif

extern "C" uint2 thrust_partitionDomains( my_dev::dev_mem<uint2> &validList,
                                          my_dev::dev_mem<uint2> &validList2, //Unsorted compacted list
                                          my_dev::dev_mem<uint>  &idList,
                                          my_dev::dev_mem<uint2> &outputKeys,
                                          my_dev::dev_mem<uint>  &outputValues,
                                          const int N,
                                          my_dev::dev_mem<uint>  &generalBuffer,
                                          const int currentOffset)
{
  thrust::device_ptr<uint2> values      = thrust::device_pointer_cast(validList.raw_p());
  thrust::device_ptr<uint>  listofones  = thrust::device_pointer_cast(idList.raw_p());
  thrust::device_ptr<uint2> outKeys     = thrust::device_pointer_cast(outputKeys.raw_p());
  thrust::device_ptr<uint>  outValues   = thrust::device_pointer_cast(outputValues.raw_p());
  thrust::plus<int> binary_op;


#if THRUST_VERSION >=  100700
  cached_allocator alloc(generalBuffer, currentOffset);

  double t1 = get_time();
  //Partition the values by in or out of domain. Result: [[outside],[inside ids]]
  thrust::device_ptr<uint2>  res = thrust::partition(thrust::cuda::par(alloc), values, values + N, isInOurDomain());
  const int remoteParticles      = (int) (res-values);
  double t2 = get_time();

  validList2.copy_devonly(validList, remoteParticles); //Copy the list before sorting, needed for internal move

  //Sort the outside our domain particles by their domain index
  //Result: [[ids domain0],[ids domain1], [ids domain2], ...]
  thrust::stable_sort(thrust::cuda::par(alloc),
                      values,
                      values + remoteParticles,
                      domainCompare());
//  hipDeviceSynchronize();

  double t3 = get_time();
  //Reduce the domains. The result is that we get per domain the number of particles
  //that will be send to that process. These are stored into the output buffers
  thrust::pair<thrust::device_ptr<uint2>,thrust::device_ptr<uint> > new_end;
  new_end = thrust::reduce_by_key(thrust::cuda::par(alloc),
                                  values,                   //inputIterator1
                                  values + remoteParticles, //InputIterator1
                                  listofones,               //InputIterator2
                                  outKeys,                  //OutputIterator1
                                  outValues,                //OutputIterator2
                                  domainCompare2(),
                                  binary_op);

#else
  //Partition the values by in or out of domain. Result: [[outside],[inside ids]]
   double t1 = get_time();
  thrust::device_ptr<uint2>  res = thrust::partition(values, values + N, isInOurDomain());
  const int remoteParticles      = (int) (res-values);
  double t2 = get_time();
  validList2.copy_devonly(validList, remoteParticles); //Copy the list before sorting, needed for internal move

  //Sort the outside our domain particles by their domain index
  //Result: [[ids domain0],[ids domain1], [ids domain2], ...]
  thrust::stable_sort(values,  values + remoteParticles, domainCompare());
  //hipDeviceSynchronize();
  double t3 = get_time();
  //Reduce the domains. The result is that we get per domain the number of particles
  //that will send to that process. These are stored into the output buffers
  thrust::pair<thrust::device_ptr<uint2>,thrust::device_ptr<uint> > new_end;
  new_end = thrust::reduce_by_key(values,                   //inputIterator1
                                  values + remoteParticles, //InputIterator1
                                  listofones,              //InputIterator2
                                  outKeys,                  //OutputIterator1
                                  outValues,                  //OutputIterator2
                                  domainCompare2(),
                                  binary_op);
#endif

  LOGF(stderr,"Sorting detail: N: %d partition: %lg sort: %lg reduce: %lg \n",remoteParticles, t2-t1,t3-t2,get_time()-t3);

  const int nValues = (int)(new_end.first  - outKeys);
  //return the number of remote particles and the number of remote domains
  return make_uint2(remoteParticles, nValues);
}


static __device__ inline int cmp_uint4(uint4 a, uint4 b) {
  if      (a.x < b.x) return -1;
  else if (a.x > b.x) return +1;
  else {
    if       (a.y < b.y) return -1;
    else  if (a.y > b.y) return +1;
    else {
      if       (a.z < b.z) return -1;
      else  if (a.z > b.z) return +1;
      return 0;
    } //end z
  }  //end y
} //end x, function



//Binary search of the key within certain bounds (cij.x, cij.y)
//Note this is the same as 'find_key'
static __device__ int find_domain(uint4 key, uint2 cij, uint4 *keys) {
  int l = cij.x;
  int r = cij.y - 1;
  while (r - l > 1) {
    int m = (r + l) >> 1;
    int cmp = cmp_uint4(keys[m], key);
    //if(cmp == 0) return m;
    if(0) {}
    else if (cmp == -1) {
      l = m;
    } else {
      r = m;
    }
  }

  //if (cmp_uint4(keys[r], key) == 0) return r;
  //return l;

  if (cmp_uint4(keys[l], key) >= 0) return l;
  return r;
}
//Check if a particles key is within the min and max boundaries
KERNEL_DECLARE(gpu_domainCheckSFCAndAssign)(int    n_bodies,
                                            int    nProcs,
                                            uint4  lowBoundary,
                                            uint4  highBoundary,
                                            uint4  *boundaryList, //The full list of boundaries
                                            uint4  *body_key,
                                            uint2  *validList,    //Valid is 1 if particle is outside domain,
                                            uint   *idList, int procId
){
  CUXTIMER("domainCheckSFCAndAssign");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if (id >= n_bodies) return;

  uint4 key = body_key[id];

  int bottom = cmp_uint4(key, lowBoundary);
  int top    = cmp_uint4(key, highBoundary);

  uint valid = 0;
  if(bottom >= 0 && top < 0)
  {
    //Inside
  }
  else
  {
    //outside
    //Search the box that this particle belongs to. Note we start at idx[1] that
    //way we get the top-end values of the domain
    uint2 cij;
    cij.x = 0; cij.y = nProcs+1;
    int domain = find_domain(key, cij, &boundaryList[1]);

    if(procId == domain) domain = domain + 1;

    valid = domain | ((1) << 31);
  }

  validList[id] = make_uint2(valid, id);
  idList[id]    = 1;
}


KERNEL_DECLARE(gpu_internalMoveSFC2) (int       n_extract,
                                  int       n_bodies,
                                  uint4  lowBoundary,
                                  uint4  highBoundary,
                                  int2       *extractList,
                                  int       *indexList,
                                  real4     *Ppos,
                                  real4     *Pvel,
                                  real4     *pos,
                                  real4     *vel,
                                  real4     *acc0,
                                  real4     *acc1,
                                  float2    *time,
                                  unsigned long long       *body_id,
                                  uint4     *body_key,
                                  float *h)
{
  CUXTIMER("internalMoveSFC2");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if(id >= n_extract) return;

  int srcIdx     = (n_bodies-n_extract) + id;


  uint4 key  = body_key[srcIdx];
  int bottom = cmp_uint4(key, lowBoundary);
  int top    = cmp_uint4(key, highBoundary);


  if((bottom >= 0 && top < 0))
  {
    int dstIdx = atomicAdd(indexList, 1);
    dstIdx     = extractList[dstIdx].y;

    //Move!
    Ppos[dstIdx] = Ppos[srcIdx];
    Pvel[dstIdx] = Pvel[srcIdx];
    pos[dstIdx]  = pos[srcIdx];
    vel[dstIdx]  = vel[srcIdx];
    acc0[dstIdx] = acc0[srcIdx];
    acc1[dstIdx] = acc1[srcIdx];
    time[dstIdx] = time[srcIdx];
    body_key[dstIdx] = body_key[srcIdx];
    body_id[dstIdx]  = body_id[srcIdx];
    h[dstIdx]     = h[srcIdx];
  }//if inside

}



KERNEL_DECLARE(gpu_extractOutOfDomainParticlesAdvancedSFC2)(
                                                       int offset,
                                                       int n_extract,
                                                       uint2 *extractList,
                                                       real4 *Ppos,
                                                       real4 *Pvel,
                                                       real4 *pos,
                                                       real4 *vel,
                                                       real4 *acc0,
                                                       real4 *acc1,
                                                       float2 *time,
                                                       unsigned long long    *body_id,
                                                       uint4 *body_key,
                                                       float *h,
                                                       bodyStruct *destination)
{
  CUXTIMER("extractOutOfDomainParticlesAdvancedSFC2");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;


#if 0
  //slowest
  if(id >= n_extract) return;

  //copy the data from a struct of arrays into a array of structs
  destination[id].Ppos = Ppos[extractList[offset+id].y];
  destination[id].Pvel = Pvel[extractList[offset+id].y];
  destination[id].pos  = pos[extractList[offset+id].y];
  destination[id].vel  = vel[extractList[offset+id].y];
  destination[id].acc0  = acc0[extractList[offset+id].y];
  destination[id].acc1  = acc1[extractList[offset+id].y];
  destination[id].time  = time[extractList[offset+id].y];
  destination[id].id    = body_id[extractList[offset+id].y];
  destination[id].key   = body_key[extractList[offset+id].y];

#elif 0
  This one does not work with 96 byte structures

  //5x faster than original (above)
  __shared__ bodyStruct shmem[128];

  if((bid * blockDim.x) >= n_extract) return;

  if(id < n_extract)
  {
	shmem[threadIdx.x].pos   = pos[extractList[offset+id].y];
	shmem[threadIdx.x].vel   = vel[extractList[offset+id].y];
    shmem[threadIdx.x].Ppos  = Ppos[extractList[offset+id].y];
    shmem[threadIdx.x].Pvel  = Pvel[extractList[offset+id].y];
    shmem[threadIdx.x].acc0  = acc0[extractList[offset+id].y];
    shmem[threadIdx.x].time  = time[extractList[offset+id].y];

    shmem[threadIdx.x].id     = body_id[extractList[offset+id].y];
    shmem[threadIdx.x].Pvel.w = h[extractList[offset+id].y];


#ifdef DO_BLOCK_TIMESTEP_EXCHANGE_MPI
    shmem[threadIdx.x].key   = body_key[extractList[offset+id].y];
    shmem[threadIdx.x].acc1  = acc1[extractList[offset+id].y];
#endif
  }
  __syncthreads();

  int startWrite  = bid * blockDim.x;
  float4 *shdata4 = (float4*)shmem;
  float4 *output  = (float4*)&destination[startWrite];


  //We have blockDim.x thread, each thread writes a float4. Compute number of items per thread-block
  //and number of loops and remaining items
  const float nThreadsPerItem = sizeof(bodyStruct) / sizeof(float4);
  const int   nItemsPerLoop   = (int)(blockDim.x / nThreadsPerItem);

  const int nExtractThisBlock = min(n_extract-startWrite, (int)blockDim.x);

  const int   nLoops          = (nExtractThisBlock/nItemsPerLoop);
  int         nExtra          = (nExtractThisBlock - nLoops*nItemsPerLoop)*nThreadsPerItem;


  int startOut = 0;
  for(int i=0; i < nLoops; i++)
  {
    output[startOut + threadIdx.x] = shdata4[threadIdx.x  + startOut]; //Write blockDim.x * float4 items
    startOut += blockDim.x;
  }

  //Write the remaining items
  if(threadIdx.x < nExtra)
  {
//    output[startOut + threadIdx.x] = shdata4[threadIdx.x  + startOut]; //Write remaining float4 items
  }

#elif 1
  //5x faster than original (above)
  __shared__ bodyStruct shmem[128];

  if((bid * blockDim.x) >= n_extract) return;

  if(id < n_extract)
  {
	shmem[threadIdx.x].pos   = pos[extractList[offset+id].y];
	shmem[threadIdx.x].vel   = vel[extractList[offset+id].y];
    shmem[threadIdx.x].Ppos  = Ppos[extractList[offset+id].y];
    shmem[threadIdx.x].Pvel  = Pvel[extractList[offset+id].y];
    shmem[threadIdx.x].acc0  = acc0[extractList[offset+id].y];
    shmem[threadIdx.x].time  = time[extractList[offset+id].y];

    shmem[threadIdx.x].id     = body_id[extractList[offset+id].y];
    shmem[threadIdx.x].Pvel.w = h[extractList[offset+id].y];


#ifdef DO_BLOCK_TIMESTEP_EXCHANGE_MPI
    shmem[threadIdx.x].key   = body_key[extractList[offset+id].y];
    shmem[threadIdx.x].acc1  = acc1[extractList[offset+id].y];
#endif
  }
  __syncthreads();

  int startWrite  = bid * blockDim.x;
  float4 *shdata4 = (float4*)shmem;
  float4 *output4 = (float4*)&destination[startWrite];


  //We have blockDim.x thread, each thread writes a float4. Compute number of items per thread-block
  //and number of loops and remaining items
  const int nExtractThisBlock = min(n_extract-startWrite, (int)blockDim.x);
  const int nFloatItems   	  = (nExtractThisBlock*sizeof(bodyStruct)) / sizeof(float4);
  const int nLoops      	  = nFloatItems / blockDim.x;

  int startOut = 0;
  for(int i=0; i < nLoops; i++)
  {
    output4[startOut + threadIdx.x] = shdata4[threadIdx.x  + startOut]; //Write blockDim.x * float4 items
    startOut += blockDim.x;
  }

  //Compute number of remaining float sized items (this requires bodyStruct to be a multiple of floats)
  const int nExtraFloats 	  = ((nExtractThisBlock*sizeof(bodyStruct)) -
		  	  	  	  	  	    (nLoops*blockDim.x*sizeof(float4))) / sizeof(float);

  float *shdata = (float*)&shdata4[nLoops*blockDim.x];
  float *output = (float*)&output4[nLoops*blockDim.x];

  //Write the remaining items
  for(int i=0; i < nExtraFloats; i+= blockDim.x)
  {
	  if(threadIdx.x + i < nExtraFloats)
	  {
		  output[i + threadIdx.x] = shdata[threadIdx.x  + i];
	  }
  }

#endif
}

KERNEL_DECLARE(gpu_insertNewParticlesSFC)(int       	 n_extract,
                                          int       	 n_insert,
										  int       	 n_oldbodies,
										  int       	 offset,
										  real4     	*Ppos,
										  real4     	*Pvel,
										  real4     	*pos,
										  real4     	*vel,
										  real4     	*acc0,
										  real4     	*acc1,
										  float2    	*time,
										  unsigned long long        *body_id,
										  uint4     	*body_key,
										  float     	*h,
										  bodyStruct 	*source)
{
  CUXTIMER("insertNewParticlesSFC");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if(id >= n_insert) return;

  //The newly added particles are added at the end of the array
  int idx = (n_oldbodies-n_extract) + id + offset;

  //copy the data from a struct of arrays into a array of structs
  pos [idx]     = source[id].pos;
  vel [idx]     = source[id].vel;
  Ppos[idx]     = source[id].Ppos;
  Pvel[idx]     = source[id].Pvel;
  acc0[idx]     = source[id].acc0;
  time[idx]     = source[id].time;
  body_id[idx]  = source[id].id;

  h[idx]        = source[id].Pvel.w;


#ifdef DO_BLOCK_TIMESTEP_EXCHANGE_MPI
  body_key[idx] = source[id].key;
  acc1[idx]     = source[id].acc1;
#endif
}


#if 0




#if 0
  Do not use the below kernels without first checking
  that the offsets are correct. There were problems with the ones
  above.

#elif 0
  //
  __shared__ bodyStruct shmem[64];
  #define WARP_SIZE2 5
  #define WARP_SIZE  32
  #define laneId (threadIdx.x & (WARP_SIZE - 1))
  #define warpId (threadIdx.x >> WARP_SIZE2)

  int startOut = 0;
  for(int j=0; j < 2; j++)
  {
    int nExtractThisBlock = min(n_extract-(bid * blockDim.x), (int)blockDim.x);
    nExtractThisBlock    -= j*64;
    nExtractThisBlock     = min(64, nExtractThisBlock);

    int readIdx = j*64+offset+id;

    __syncthreads();
    if(warpId < 2)
    {
      if(readIdx < n_extract)
      {
        readIdx                  = extractList[readIdx].y;
        shmem[threadIdx.x].Ppos  = Ppos[readIdx];
        shmem[threadIdx.x].Pvel  = Pvel[readIdx];
        shmem[threadIdx.x].pos   = pos[readIdx];
        shmem[threadIdx.x].vel   = vel[readIdx];
      }
    }
    else
    {
      if(readIdx-64 < n_extract)
      {
        readIdx                       = extractList[readIdx- 64].y;
        shmem[threadIdx.x - 64].acc0  = acc0[readIdx];
        shmem[threadIdx.x - 64].acc1  = acc1[readIdx];
        shmem[threadIdx.x - 64].time  = time[readIdx];
        shmem[threadIdx.x - 64].id    = body_id[readIdx];
        shmem[threadIdx.x - 64].key   = body_key[readIdx];
      }
    }
    __syncthreads();


    int startWrite  = bid * blockDim.x;
    float4 *shdata4 = (float4*)shmem;
    float4 *output  = (float4*)&destination[startWrite];

    //We have blockDim.x thread, each thread writes a float4. Compute number of items per thread-block
    //and number of loops and remaining items
    const float nThreadsPerItem = sizeof(bodyStruct) / sizeof(float4);
    const int   nItemsPerLoop   = (int)(blockDim.x / nThreadsPerItem);

    const int   nLoops          = (nExtractThisBlock/nItemsPerLoop);
    const int   nExtra          = (nExtractThisBlock - nLoops*nItemsPerLoop)*nThreadsPerItem;


  #pragma unroll
    for(int i=0; i < nLoops; i++)
    {
      output[startOut + threadIdx.x] = shdata4[threadIdx.x  + i*blockDim.x]; //Write first blockDim.x * float4 items
      startOut += blockDim.x;
    }

    //Write the remaining items
    if(threadIdx.x < nExtra)
    {
      output[startOut + threadIdx.x] = shdata4[threadIdx.x  + nLoops*blockDim.x]; //Write first blockDim.x * float4 items
    }
  } //for j

#elif 0

//Comparable to one below

  __shared__ bodyStruct shmem[32];
  #define WARP_SIZE2 5
  #define WARP_SIZE  32
  #define laneId (threadIdx.x & (WARP_SIZE - 1))
  #define warpId (threadIdx.x >> WARP_SIZE2)

  int startWrite  = bid * blockDim.x;
  float4 *shdata4 = (float4*)shmem;
  float4 *output  = (float4*)&destination[startWrite];
  int startOut    = 0;

  float4 temp1;
  float4 temp2;
  uint4  temp3;
  float2 temp4;
  int    temp5;

  for(int loop=0; loop < 4; loop++) //4 = 128 threads / 32 items
  {
    int readIdx = (bid * blockDim.x) + loop*32 + laneId + offset;
    //We only read as much as we can write in two transaction
    readIdx = min(readIdx, n_extract-1);

    if(warpId == 0)
    {
      temp1 = pos[extractList[readIdx].y];
      temp2 = Ppos[extractList[readIdx].y];
      temp3 = body_key[extractList[readIdx].y];
    }
    else if (warpId == 1)
    {
      temp1 = Pvel[extractList[readIdx].y];
      temp2 = vel[extractList[readIdx].y];
    }
    else if (warpId == 2)
    {
      temp1 = acc0[extractList[readIdx].y];
      temp2 = acc1[extractList[readIdx].y];
    }
    else if (warpId == 3)
    {
      temp5   = body_id[extractList[readIdx].y];
      temp4 = time[extractList[readIdx].y];
    }

    __syncthreads();

    if(warpId == 0)
    {
      shmem[laneId].pos = temp1;
      shmem[laneId].Ppos = temp2;
      shmem[laneId].key = temp3;
    }
    else if (warpId == 1)
    {
      shmem[laneId].Pvel = temp1;
      shmem[laneId].vel = temp2;
    }
    else if (warpId == 2)
    {
      shmem[laneId].acc0 = temp1;
      shmem[laneId].acc1 = temp2;
    }
    else if (warpId == 3)
    {
      shmem[laneId].id   = temp5;
      shmem[laneId].time = temp4;
    }
    __syncthreads();

    for(int i=0; i < 2; i++) //2 is 32 items / 16 items write per 128 threads
    {
      if(startOut + threadIdx.x < (8*n_extract)) //8* sincce 8 float4 in bodystruct
        output[startOut + threadIdx.x] = shdata4[threadIdx.x  + i*blockDim.x]; //Write first blockDim.x * float4 items
      startOut += blockDim.x;
    }

  }
#elif 0
//Second fastest
  __shared__ bodyStruct shmem[32];
  #define WARP_SIZE2 5
  #define WARP_SIZE  32
  #define laneId (threadIdx.x & (WARP_SIZE - 1))
  #define warpId (threadIdx.x >> WARP_SIZE2)

  int startWrite  = bid * blockDim.x;
  float4 *shdata4 = (float4*)shmem;
  float4 *output  = (float4*)&destination[startWrite];
  int startOut    = 0;

  for(int loop=0; loop < 4; loop++) //4 = 128 threads / 32 items
  {
    int readIdx = (bid * blockDim.x) + loop*32 + laneId  + offset;
    //We only read as much as we can write in two transaction
    readIdx = min(readIdx, n_extract-1);

    __syncthreads();
    if(warpId == 0)
    {
      shmem[laneId].pos = pos[extractList[readIdx].y];
      shmem[laneId].Ppos = Ppos[extractList[readIdx].y];
      shmem[laneId].key = body_key[extractList[readIdx].y];
    }
    else if (warpId == 1)
    {
      shmem[laneId].Pvel = Pvel[extractList[readIdx].y];
      shmem[laneId].vel = vel[extractList[readIdx].y];
    }
    else if (warpId == 2)
    {
      shmem[laneId].acc0 = acc0[extractList[readIdx].y];
      shmem[laneId].acc1 = acc1[extractList[readIdx].y];
    }
    else if (warpId == 3)
    {
      shmem[laneId].id   = body_id[extractList[readIdx].y];
      shmem[laneId].time = time[extractList[readIdx].y];
    }

    __syncthreads();

    for(int i=0; i < 2; i++) //2 is 32 items / 16 items write per 128 threads
    {
      if(startOut + threadIdx.x < (8*n_extract)) //8* sincce 8 float4 in bodystruct
        output[startOut + threadIdx.x] = shdata4[threadIdx.x  + i*blockDim.x]; //Write first blockDim.x * float4 items
      startOut += blockDim.x;
    }

  }

#endif






KERNEL_DECLARE(gpu_internalMoveSFC) (int       n_extract,
                                  int       n_bodies,
                                  uint4  lowBoundary,
                                  uint4  highBoundary,
                                  int       *extractList,
                                  int       *indexList,
                                  real4     *Ppos,
                                  real4     *Pvel,
                                  real4     *pos,
                                  real4     *vel,
                                  real4     *acc0,
                                  real4     *acc1,
                                  float2    *time,
                                  unsigned long long        *body_id,
                                  uint4     *body_key
          )
{
  CUXTIMER("internalMoveSFC");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if(id >= n_extract) return;

  int srcIdx     = (n_bodies-n_extract) + id;


  uint4 key  = body_key[srcIdx];
  int bottom = cmp_uint4(key, lowBoundary);
  int top    = cmp_uint4(key, highBoundary);


  if((bottom >= 0 && top < 0))
  {
    int dstIdx = atomicAdd(indexList, 1);
    dstIdx     = extractList[dstIdx];

    //Move!
    Ppos[dstIdx] = Ppos[srcIdx];
    Pvel[dstIdx] = Pvel[srcIdx];
    pos[dstIdx]  = pos[srcIdx];
    vel[dstIdx]  = vel[srcIdx];
    acc0[dstIdx] = acc0[srcIdx];
    acc1[dstIdx] = acc1[srcIdx];
    time[dstIdx] = time[srcIdx];
    body_key[dstIdx] = body_key[srcIdx];
    body_id[dstIdx]  = body_id[srcIdx];
  }//if inside

}
#endif

#if 0
KERNEL_DECLARE(gpu_internalMove)(int       n_extract,
                                        int       n_bodies,
                                        double4  xlow,
                                        double4  xhigh,
                                        int       *extractList,
                                        int       *indexList,
                                        real4     *Ppos,
                                        real4     *Pvel,
                                        real4     *pos,
                                        real4     *vel,
                                        real4     *acc0,
                                        real4     *acc1,
                                        float2    *time,
                                        int       *body_id)
{
  CUXTIMER("internalMove");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if(id >= n_extract) return;

  int srcIdx     = (n_bodies-n_extract) + id;
  real4 testpos  = Ppos[srcIdx];

  if(isinbox(testpos, xlow, xhigh))
  {
    int dstIdx = atomicAdd(indexList, 1);
    dstIdx     = extractList[dstIdx];

    //Move!
    Ppos[dstIdx] = Ppos[srcIdx];
    Pvel[dstIdx] = Pvel[srcIdx];
    pos[dstIdx]  = pos[srcIdx];
    vel[dstIdx]  = vel[srcIdx];
    acc0[dstIdx] = acc0[srcIdx];
    acc1[dstIdx] = acc1[srcIdx];
    time[dstIdx] = time[srcIdx];
    body_id[dstIdx] = body_id[srcIdx];
  }//if isinbox

}



//Check if a particles key is within the min and max boundaries
KERNEL_DECLARE(gpu_domainCheckSFC)(int    n_bodies,
                               uint4  lowBoundary,
                               uint4  highBoundary,
                               uint4  *body_key,
                               int    *validList    //Valid is 1 if particle is outside domain
){
  CUXTIMER("domainCheckSFC");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if (id >= n_bodies) return;

  uint4 key = body_key[id];

  int bottom = cmp_uint4(key, lowBoundary);
  int top    = cmp_uint4(key, highBoundary);

  int valid = 0;
  if(bottom >= 0 && top < 0)
  {
    //INside
  }
  else
  {
    //    outside
    valid = 1;
  }
  validList[id] = id | ((valid) << 31);
}
#endif

#if 0
KERNEL_DECLARE(gpu_extractSampleParticlesSFC)(int     n_bodies,
                                              int     nSamples,
                                              float   sample_freq,
                                              uint4  *body_pos,
                                              uint4  *samplePosition
){
  CUXTIMER("extractSampleParticles");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  if(id >= nSamples) return;

  int idx  = (int)(id*sample_freq);
  if  (idx >= n_bodies) return;

  samplePosition[id] =  body_pos[idx];
}
#endif
