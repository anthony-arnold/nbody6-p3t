#include "hip/hip_runtime.h"
#include "bonsai.h"

#include "support_kernels.cu"
#include <stdio.h>

#include "../profiling/bonsai_timing.h"
PROF_MODULE(build_tree);

#if 0
__device__ float atomicMin(float* address, float val)
{
  int* addressI = (int*)address;
  int old = *addressI, prev;
  do {
    prev = old;
    old  = atomicCAS(addressI, prev, __float_as_int(::fminf(val, __int_as_float(prev))));
  } while (prev != old);
  return __int_as_float(old);
}
__device__ float atomicMax(float* address, float val)
{
  int* addressI = (int*)address;
  int old = *addressI, prev;
  do {
    prev = old;
    old  = atomicCAS(addressI, prev, __float_as_int(::fmaxf(val, __int_as_float(prev))));
  } while (prev != old);
  return __int_as_float(old);
}
#endif


//TODO merge boundaryReduction and groupBoundary reduction
//into a single kernel. And remove the 256 constant make it blockDim.x
KERNEL_DECLARE(gpu_boundaryReduction)(const int         n_particles,
                                            real4      *positions,
                                            float3     *output_min,
                                            float3     *output_max)
{
  CUXTIMER("boundaryReduction");
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;

  volatile __shared__ float3 shmem[512];
  float3 r_min = make_float3(+1e10f, +1e10f, +1e10f);
  float3 r_max = make_float3(-1e10f, -1e10f, -1e10f);

  volatile float3 *sh_rmin = (float3*)&shmem [ 0];
  volatile float3 *sh_rmax = (float3*)&shmem[256];
  sh_rmin[tid].x = r_min.x; sh_rmin[tid].y = r_min.y; sh_rmin[tid].z = r_min.z;
  sh_rmax[tid].x = r_max.x; sh_rmax[tid].y = r_max.y; sh_rmax[tid].z = r_max.z;

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  const int blockSize   = blockDim.x;
//   unsigned int tid      = threadIdx.x;
  unsigned int i        = blockIdx.x*(blockSize*2) + threadIdx.x;
  unsigned int gridSize = blockSize*2*gridDim.x;

  real4 pos;
  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridSize).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  //based on reduce6 example
  while (i < n_particles) {
    if (i             < n_particles)
    {
      pos = positions[i];
      r_min.x = fminf(pos.x, r_min.x); r_min.y = fminf(pos.y, r_min.y); r_min.z = fminf(pos.z, r_min.z);
      r_max.x = fmaxf(pos.x, r_max.x); r_max.y = fmaxf(pos.y, r_max.y); r_max.z = fmaxf(pos.z, r_max.z);
    }
    if (i + blockSize < n_particles)
    {
      pos = positions[i + blockSize];
      r_min.x = fminf(pos.x, r_min.x); r_min.y = fminf(pos.y, r_min.y); r_min.z = fminf(pos.z, r_min.z);
      r_max.x = fmaxf(pos.x, r_max.x); r_max.y = fmaxf(pos.y, r_max.y); r_max.z = fmaxf(pos.z, r_max.z);
    }
    i += gridSize;
  }

  sh_rmin[tid].x = r_min.x; sh_rmin[tid].y = r_min.y; sh_rmin[tid].z = r_min.z;
  sh_rmax[tid].x = r_max.x; sh_rmax[tid].y = r_max.y; sh_rmax[tid].z = r_max.z;

  __syncthreads();
  // do reduction in shared mem  
  if(blockDim.x >= 512) if (tid < 256) {sh_MinMax(tid, tid + 256, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
  if(blockDim.x >= 256) if (tid < 128) {sh_MinMax(tid, tid + 128, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
  if(blockDim.x >= 128) if (tid < 64)  {sh_MinMax(tid, tid + 64,  &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();

  if (tid < 32) 
  {
    sh_MinMax(tid, tid + 32, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid + 16, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  8, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  4, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  2, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  1, &r_min, &r_max, sh_rmin,sh_rmax);
  }

  // write result for this block to global mem
  if (tid == 0)
  {
    //Compiler doesnt allow: volatile float3 = float3
    output_min[bid].x = sh_rmin[0].x; output_min[bid].y = sh_rmin[0].y; output_min[bid].z = sh_rmin[0].z;
    output_max[bid].x = sh_rmax[0].x; output_max[bid].y = sh_rmax[0].y; output_max[bid].z = sh_rmax[0].z;
//    atomicMin(&output_min[0].x, sh_rmin[0].x);
//    atomicMin(&output_min[0].y, sh_rmin[0].y);
//    atomicMin(&output_min[0].z, sh_rmin[0].z);
//    atomicMax(&output_max[0].x, sh_rmax[0].x);
//    atomicMax(&output_max[0].y, sh_rmax[0].y);
//    atomicMax(&output_max[0].z, sh_rmax[0].z);
  }

}

//Get the domain size, by taking into account the group size
KERNEL_DECLARE(gpu_boundaryReductionGroups)(const int n_groups,
                                                   real4      *positions,
                                                   real4      *sizes,
                                                   float3     *output_min,
                                                   float3     *output_max)
{
  CUXTIMER("boundaryReductionGroups");
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  //const uint idx = bid * blockDim.x + tid;

  volatile __shared__ float3 shmem[512];
  float3 r_min = make_float3(+1e10f, +1e10f, +1e10f);
  float3 r_max = make_float3(-1e10f, -1e10f, -1e10f);

  volatile float3 *sh_rmin = (float3*)&shmem [ 0];
  volatile float3 *sh_rmax = (float3*)&shmem[256];
  sh_rmin[tid].x = r_min.x; sh_rmin[tid].y = r_min.y; sh_rmin[tid].z = r_min.z;
  sh_rmax[tid].x = r_max.x; sh_rmax[tid].y = r_max.y; sh_rmax[tid].z = r_max.z;

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  const int blockSize   = blockDim.x;
//   unsigned int tid      = threadIdx.x;
  unsigned int i        = blockIdx.x*(blockSize*2) + threadIdx.x;
  unsigned int gridSize = blockSize*2*gridDim.x;

  real4 pos;
  real4 size;
  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridSize).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  //based on reduce6 example
  while (i < n_groups) {
    if (i             < n_groups)
    {
      pos = positions[i];
      size = sizes[i];
      r_min.x = fminf(pos.x-size.x, r_min.x);
      r_min.y = fminf(pos.y-size.y, r_min.y);
      r_min.z = fminf(pos.z-size.z, r_min.z);
      r_max.x = fmaxf(pos.x+size.x, r_max.x);
      r_max.y = fmaxf(pos.y+size.y, r_max.y);
      r_max.z = fmaxf(pos.z+size.z, r_max.z);
    }
    if (i + blockSize < n_groups)
    {
      pos = positions[i + blockSize];
      size = sizes[i + blockSize];
      r_min.x = fminf(pos.x-size.x, r_min.x);
      r_min.y = fminf(pos.y-size.y, r_min.y);
      r_min.z = fminf(pos.z-size.z, r_min.z);
      r_max.x = fmaxf(pos.x+size.x, r_max.x);
      r_max.y = fmaxf(pos.y+size.y, r_max.y);
      r_max.z = fmaxf(pos.z+size.z, r_max.z);
    }
    i += gridSize;
  }

  sh_rmin[tid].x = r_min.x; sh_rmin[tid].y = r_min.y; sh_rmin[tid].z = r_min.z;
  sh_rmax[tid].x = r_max.x; sh_rmax[tid].y = r_max.y; sh_rmax[tid].z = r_max.z;

  __syncthreads();
  // do reduction in shared mem  
  if(blockDim.x >= 512) if (tid < 256) {sh_MinMax(tid, tid + 256, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
  if(blockDim.x >= 256) if (tid < 128) {sh_MinMax(tid, tid + 128, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
  if(blockDim.x >= 128) if (tid < 64)  {sh_MinMax(tid, tid + 64,  &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();

  if (tid < 32) 
  {
    sh_MinMax(tid, tid + 32, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid + 16, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  8, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  4, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  2, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  1, &r_min, &r_max, sh_rmin,sh_rmax);
  }


  // write result for this block to global mem
  if (tid == 0)
  {
    //Compiler doesnt allow: volatile float3 = float3
    output_min[bid].x = sh_rmin[0].x; output_min[bid].y = sh_rmin[0].y; output_min[bid].z = sh_rmin[0].z;
    output_max[bid].x = sh_rmax[0].x; output_max[bid].y = sh_rmax[0].y; output_max[bid].z = sh_rmax[0].z;
  }

}

//#define EXACT_KEY

KERNEL_DECLARE(cl_build_key_list)(uint4  *body_key,
                                            real4  *body_pos,
                                            int   n_bodies,
                                            real4  corner) {
  
  CUXTIMER("cl_build_key_list");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;
  
  if (id > n_bodies) return;

  real4 pos = body_pos[id];

  int4 crd;
  
  real domain_fac = corner.w;
  
  #ifndef EXACT_KEY
    crd.x = (int)roundf(__fdividef((pos.x - corner.x), domain_fac));
    crd.y = (int)roundf(__fdividef((pos.y - corner.y) , domain_fac));
    crd.z = (int)roundf(__fdividef((pos.z - corner.z) , domain_fac));
  #else            
    crd.x = (int)((pos.x - corner.x) / domain_fac);
    crd.y = (int)((pos.y - corner.y) / domain_fac);
    crd.z = (int)((pos.z - corner.z) / domain_fac);
  #endif

  uint4 key = get_key(crd);

  if (id == n_bodies) key = make_uint4(0xFFFFFFFF, 0xFFFFFFFF, 0, 0);

  key.w        = id;
  body_key[id] = key;
}
  

KERNEL_DECLARE(cl_build_valid_list)(int n_bodies,
                                               int level,
                                               uint4  *body_key,
                                               uint *valid_list,
                                               const uint *workToDo) {
  if (0 == *workToDo) return;
//                                                uint2 *test_key_data) {
  CUXTIMER("cl_build_valid_list");
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  const uint id  = bid * blockDim.x + tid;
  const uint4 key_F = {0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF};
  
  if (id >= n_bodies) return;   // >=   since the last particle is extra boudnary particle
  
  uint4 mask = get_mask(level);
  mask.x = mask.x | ((uint)1 << 30) | ((uint)1 << 31);

  uint4 key_m;
  uint4 key_c    = body_key[id];
  uint4 key_p;

  if (id == 0)
  {
    key_m = key_F;
  }
  else
  {
    key_m = body_key[id-1];
  }

  if((id+1) <  n_bodies) //The last particle gets a different key to compare with
  {
    key_p = body_key[id+1];
  }
  else
    key_p = make_uint4(0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF);


  int valid0 = 0;
  int valid1 = 0;

  if (cmp_uint4(key_c, key_F) != 0) {
    key_c.x = key_c.x & mask.x;
    key_c.y = key_c.y & mask.y;
    key_c.z = key_c.z & mask.z;

    key_p.x = key_p.x & mask.x;
    key_p.y = key_p.y & mask.y;
    key_p.z = key_p.z & mask.z;

    key_m.x = key_m.x & mask.x;
    key_m.y = key_m.y & mask.y;
    key_m.z = key_m.z & mask.z;

    valid0 = abs(cmp_uint4(key_c, key_m));
    valid1 = abs(cmp_uint4(key_c, key_p));
  }

   valid_list[id*2]   = id | ((valid0) << 31);
   valid_list[id*2+1] = id | ((valid1) << 31);

}


//////////////////////////////
//////////////////////////////
//////////////////////////////
static __device__ uint retirementCountBuildNodes = 0;

KERNEL_DECLARE(cl_build_nodes)(uint level,
                             uint  *compact_list_len,
                             uint  *level_offset,
                             uint  *last_level,
                             uint2 *level_list,
                             uint  *compact_list,
                             uint4 *bodies_key,
                             uint4 *node_key,
                             uint  *n_children,
                             uint2 *node_bodies){

  CUXTIMER("cl_build_nodes");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  uint n = (*compact_list_len)/2;
  uint offset = *level_offset;

  //We reuse last_level as indicator if we are allowed to create LEAF nodes
  bool minLevelReached = (int)*last_level;

  for (; id < n; id += gridDim.x * gridDim.y * blockDim.x)
  {
    uint  bi   = compact_list[id*2];
    uint  bj   = compact_list[id*2+1] + 1;
  
    uint4 key  = bodies_key[bi];
    uint4 mask = get_mask(level);
    key = make_uint4(key.x & mask.x, key.y & mask.y, key.z & mask.z, 0); 

    node_bodies[offset+id] = make_uint2(bi | (level << BITLEVELS), bj);
    node_key   [offset+id] = key;
    n_children [offset+id] = 0;
  
//    if ((int)level > (int)(LEVEL_MIN - 1))
    if(minLevelReached)
      if (bj - bi <= NLEAF)                            //Leaf can only have NLEAF particles, if its more there will be a split
        for (int i = bi; i < bj; i++)
          bodies_key[i] = make_uint4(0xFFFFFFFF,0xFFFFFFFF,0xFFFFFFFF,0xFFFFFFFF); //sets the key to FF to indicate the body is used
  }

  //
  // PHASE 2: Last block updates level list and offset
  //

  int numBlocks = gridDim.x * gridDim.y;
  if (numBlocks > 1)
  {
    __shared__ bool amLast;

    // Thread 0 takes a ticket
    if( tid==0 )
    {
      unsigned int ticket = atomicInc(&retirementCountBuildNodes, numBlocks);
      // If the ticket ID is equal to the number of blocks, we are the last block!
      amLast = (ticket == numBlocks-1);
    }
    __syncthreads();

    // The last block sums the results of all other blocks
    if( amLast && tid == 0)
    {           
      level_list[level] = (n > 0) ? make_uint2(offset, offset + n) : make_uint2(0, 0);
      *level_offset = offset + n;


      //Set last_level to a value to indicate we are now allowed to make
      //leafs. It will later be overwritten to indicate the final level
      if(n > START_LEVEL_MIN_NODES){
          *last_level = 1;
      }

      if ((level > 0) && (n <= 0) && (level_list[level - 1].x > 0))
        *last_level = level;

      // reset retirement count so that next run succeeds
      retirementCountBuildNodes = 0; 
    }
  }
}


//////////////////////////////
//////////////////////////////
//////////////////////////////


KERNEL_DECLARE(cl_link_tree)(int n_nodes,
                            uint *n_children,
                            uint2 *node_bodies,
                            real4 *bodies_pos,
                            real4 corner,
                            uint2 *level_list,
                            uint* valid_list,
                            uint4 *node_keys,
                            uint4 *bodies_key,
                            uint  levelMin) {

  CUXTIMER("cl_link_tree");
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;
  
  if (id >= n_nodes) return;

  uint2 bij  = node_bodies[id];
  uint level = (bij.x &  LEVELMASK) >> BITLEVELS;
  uint bi    =  bij.x & ILEVELMASK;
  uint bj    =  bij.y;

  real4 pos  = bodies_pos[bi];
  int4 crd;
  real domain_fac = corner.w;

  #ifndef EXACT_KEY
    crd.x = (int)roundf(__fdividef((pos.x - corner.x), domain_fac));
    crd.y = (int)roundf(__fdividef((pos.y - corner.y) , domain_fac));
    crd.z = (int)roundf(__fdividef((pos.z - corner.z) , domain_fac));
  #else            
    crd.x = (int)((pos.x - corner.x) / domain_fac);
    crd.y = (int)((pos.y - corner.y) / domain_fac);
    crd.z = (int)((pos.z - corner.z) / domain_fac);
  #endif


  uint4 key = get_key(crd);


  /********* accumulate children *****/
  
  uint4 mask = get_mask(level - 1);
  key = make_uint4(key.x & mask.x, key.y & mask.y,  key.z & mask.z, 0); 

  uint2 cij;

  
  if(id > 0)
    cij = level_list[level-1];

  int ci;
  //JB, modified this since we don't use textures in find_key,
  //the function will fail because out of bound memory access when id==0
  if(id > 0)
    ci = find_key(key, cij, node_keys);
  else
    ci = 0;

  //ci now points to the node that is the parent, was used in previous group method
  //parent_id_list[id] = ci;

  mask = get_imask(mask);
  key = make_uint4(key.x | mask.x, key.y | mask.y, key.z | mask.z, 0);
  if (id > 0)   
    atomicAdd(&n_children[ci], (1 << 28));

  key = get_key(crd);
  mask = get_mask(level);
  key = make_uint4(key.x & mask.x, key.y & mask.y, key.z & mask.z, 0); 

  /********* store the 1st child *****/

  cij = level_list[level+1];
  int cj = -1;

  cj = find_key(key, cij, node_keys);

  atomicOr(&n_children[id], cj); //Atomic since multiple threads can work on this

  uint valid =  id | (uint)(0 << 31); 

  
  if ((int)level > (int)(levelMin))
    if ((bj - bi) <= NLEAF)    
      valid = id | (uint)(1 << 31);   //Distinguish leaves and nodes

 valid_list[id] = valid; //If valid its a leaf otherwise a node
}

//Determines which level of node starts at which offset
KERNEL_DECLARE(gpu_build_level_list)(const int    n_nodes,
                                 const int    n_leafs,
                                       uint  *leafsIdxs,
                                       uint2 *node_bodies,
                                       uint  *valid_list)
{
  CUXTIMER("build_level_list");
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  const uint id  = bid * blockDim.x + tid;
  
  if (id >= n_nodes-n_leafs) return;

  const int nodeID = leafsIdxs[id+n_leafs];   //Get the idx into the node_bodies array

  int level_c, level_m, level_p;


  uint2 bij   = node_bodies[leafsIdxs[id+n_leafs]];    //current non-leaf
  level_c     = (bij.x &  LEVELMASK) >> BITLEVELS;

  if((id+1) < (n_nodes-n_leafs))        //The last node gets a default level
  {
    bij         = node_bodies[leafsIdxs[id+1+n_leafs]]; //next non-leaf
    level_p     = (bij.x &  LEVELMASK) >> BITLEVELS;
  }
  else
    level_p     = MAXLEVELS+5;  //Last is always an end

  //Compare level with the node before and node after
  if(nodeID == 0)
  {
    level_m = -1;    
  }
  else
  {
    bij         = node_bodies[ leafsIdxs[id-1+n_leafs]]; //Get info of previous non-leaf node
    level_m     =  (bij.x &  LEVELMASK) >> BITLEVELS;   
  }

  int valid0 = 0;
  int valid1 = 0;

  valid0 = (level_c != level_m) << 31 | (id+n_leafs);
  valid1 = (level_c != level_p) << 31 | (id+n_leafs);

  valid_list[id*2]   = valid0;
  valid_list[id*2+1] = valid1;

} //end build_level_list

#if 0
//Old version based on distance

//Finds nodes/leafs that will become groups
//After executions valid_list contains the 
//valid nodes/leafs that form groups
KERNEL_DECLARE(build_group_list2)(int    n_particles,
                                             uint  *validList,
                                             real4  *bodies_pos,
                                             const float DIST,
                                             int   *node_level_list,
                                             int   treeDepth)
{
  CUXTIMER("build_group_list2");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint idx = bid * blockDim.x + tid;

  __shared__ int shmem[128];

  //Compact the node_level_list
  if(bid == 0)
  {
    if(threadIdx.x < (MAXLEVELS*2))
    {
      shmem[threadIdx.x] = node_level_list[threadIdx.x];
    }

    __syncthreads(); //Can most likely do without since its one warp

    //Only selection writes
    if(threadIdx.x < MAXLEVELS)
    {
      node_level_list[threadIdx.x]  = shmem[threadIdx.x*2];
      if(threadIdx.x == treeDepth-1)
          node_level_list[threadIdx.x] = shmem[threadIdx.x*2-1]+1;
    }
  }//if bid == 0
  //end compact node level list

  //Note that we do not include the final particle
  //Since there is no reason to check it
  if (idx >= n_particles) return;

  //Get the current 
  float4 curPos, nexPos, prevPos;

  curPos  =  bodies_pos[idx];

  //Have to check the first and last to prevent out of bound access
  if(idx+1 == n_particles)
    nexPos  =  curPos;
  else
    nexPos = bodies_pos[idx+1];

  if(idx == 0)
    prevPos = curPos;
  else
    prevPos =  bodies_pos[idx-1];

  //Compute geometrical distance
  float dsPlus = ((curPos.x-nexPos.x)*(curPos.x-nexPos.x)) + 
                 ((curPos.y-nexPos.y)*(curPos.y-nexPos.y)) + 
                 ((curPos.z-nexPos.z)*(curPos.z-nexPos.z));

  float dsMin = ((curPos.x-prevPos.x)*(curPos.x-prevPos.x)) + 
                ((curPos.y-prevPos.y)*(curPos.y-prevPos.y)) + 
                ((curPos.z-prevPos.z)*(curPos.z-prevPos.z));

  //Multiples of the preferred group size are _always_ valid
  int validStart = ((idx     % NCRIT) == 0);
  int validEnd   = (((idx+1) % NCRIT) == 0);

  //The extra possible split(s) if the distance between two particles is too large
  if(dsPlus > DIST) validEnd     = 1;
  if(dsMin  > DIST) validStart   = 1;
  
  //Last particle is always the end, n_particles dont have to be a multiple of NCRIT
  //so this is required
  if(idx+1 == n_particles) validEnd = 1;

  //Set valid
  validList[2*idx + 0] = (idx)   | (uint)(validStart << 31);
  validList[2*idx + 1] = (idx+1) | (uint)(validEnd   << 31);    
}

#else
//New version based on top levels of the tree, uses top nodes/leafs which boundaries
//will become groups. After executions valid_list contains the valid nodes/leafs that form groups
KERNEL_DECLARE(build_group_list2)(const int   n_particles,
                                  uint       *validList,
                                  const uint2 startLevelBeginEnd,
                                  uint2      *node_bodies,
                                  int        *node_level_list,
                                  int         treeDepth)
{
  CUXTIMER("build_group_list2");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint idx = bid * blockDim.x + tid;

  __shared__ int shmem[128];

  //Compact the node_level_list. From begin-end positions to just begin positions
  if(bid == 0)
  {
    if(threadIdx.x < (MAXLEVELS*2))
    {
      shmem[threadIdx.x] = node_level_list[threadIdx.x];
    }

    __syncthreads(); //Can most likely do without since its one warp if MAXLEVELS < 32

    //Only selection writes
    if(threadIdx.x < MAXLEVELS)
    {
      node_level_list[threadIdx.x]  = shmem[threadIdx.x*2];
      if(threadIdx.x == treeDepth-1)
          node_level_list[threadIdx.x] = shmem[threadIdx.x*2-1]+1;
    }
  }//if bid == 0
  //end compact node level list

  //We do not include the final particle, since there is no reason to check it
  if (idx >= n_particles) return;

  //Now we get some info from tree-structure for the groups
  //Note that we do NOT include the last groups since it only sets
  //the final particle to invalid, which we will do by default anyway
  //this way we save a check on particle boundary

  //Use the end-indices of all tree-nodes above our minimum level
  if (idx < startLevelBeginEnd.y-1) //THe -1 to prevent last node
  {
    const uint2 bij          =  node_bodies[idx];
    const uint lastChild     =  bij.y;   

    //Set the boundaries, start and end 
    validList[2*lastChild - 1]  = (lastChild)   | (uint)(1 << 31);
    validList[2*lastChild]      = (lastChild)   | (uint)(1 << 31);
  }

  //Multiples of the preferred group size are _always_ valid
  int validStart = ((idx     % NCRIT) == 0);
  int validEnd   = (((idx+1) % NCRIT) == 0);
  
  //Last particle is always the end, n_particles don't have
  //to be a multiple of NCRIT so this is required
  if(idx+1 == n_particles) validEnd = 1;

  //Set valid, note only set it if we write something valid
  //otherwise we might overwrite the settings from the coarse group
  if(validStart) validList[2*idx + 0] = (idx)   | (uint)(validStart << 31);
  if(validEnd)   validList[2*idx + 1] = (idx+1) | (uint)(validEnd   << 31);    
}
#endif
 
//Store per particle the group id it belongs to
//and the start and end particle number of the groups  
KERNEL_DECLARE(store_group_list)(int    n_particles,
                                 int n_groups,
                                 uint  *validList,
                                 uint  *body2group_list,
                                 uint2 *group_list){
  CUXTIMER("store_group_list");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  //uint idx = bid * blockDim.x + tid;
  
  if(bid >= n_groups) return;

  int start = validList[2*bid];
  int end   = validList[2*bid+1];

  if((start + tid) < end)
  {
    body2group_list[start + tid] = bid;
  }

  if(tid == 0)
  {
     group_list[bid] = make_uint2(start,end);
  }
}




