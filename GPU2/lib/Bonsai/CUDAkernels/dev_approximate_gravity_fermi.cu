#include "hip/hip_runtime.h"
// #include "support_kernels.cu"
#include <stdio.h>
#include "../profiling/bonsai_timing.h"
PROF_MODULE(dev_approximate_gravity);

#include "node_specs.h"

#ifdef WIN32
#define M_PI        3.14159265358979323846264338328
#endif

__forceinline__ __device__ float Wkernel(const float q)
{
  const float sigma = 8.0f/M_PI;

  const float qm = 1.0f - q;
  const float f1 = sigma * (1.0f + (-6.0f)*q*q*qm);
  const float f2 = sigma * 2.0f*qm*qm*qm;

  return fmaxf(0.0f, fminf(f1, f2));
}

__forceinline__ __device__ float interact(
    const float3 ipos,
    const float  h,
    const float  hinv,
    const float3 jpos,
    const float  jmass)
{
  const float3 dr = make_float3(jpos.x - ipos.x, jpos.y - ipos.y, jpos.z - ipos.z);
  const float  r2 = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;
  if (r2 >= h*h) return 0.0f;
  const float q  = sqrtf(r2) * hinv;
  const float hinv3 = hinv*hinv*hinv;

  return jmass * Wkernel(q) * hinv3;
}


/***
**** --> prefix calculation via Horn(2005) data-parallel algoritm
***/
#define BTEST(x) (-(int)(x))
template<int DIM2>
__device__ int calc_prefix(int N, int* prefix_in, int tid) {
  int x, y = 0;

  const int DIM = 1 << DIM2;
  
  for (int p = 0; p < N; p += DIM) {
    int *prefix = &prefix_in[p];

    x = prefix[tid -  1]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  1); __syncthreads();
    x = prefix[tid -  2]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  2); __syncthreads();
    x = prefix[tid -  4]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  4); __syncthreads();
    x = prefix[tid -  8]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  8); __syncthreads();
    x = prefix[tid - 16]; __syncthreads(); prefix[tid] += x & BTEST(tid >= 16); __syncthreads();
    if (DIM2 >= 6) {x = prefix[tid - 32]; __syncthreads(); prefix[tid] += x & BTEST(tid >= 32); __syncthreads();}
    if (DIM2 >= 7) {x = prefix[tid - 64]; __syncthreads(); prefix[tid] += x & BTEST(tid >= 64); __syncthreads();}
    if (DIM2 >= 8) {x = prefix[tid -128]; __syncthreads(); prefix[tid] += x & BTEST(tid >=128); __syncthreads();}
    

    prefix[tid] += y;
    __syncthreads();

    y = prefix[DIM-1];
    __syncthreads();
  }

  return y;
} 

template<int DIM2>
__device__ int calc_prefix(int* prefix, int tid, int value) {
  int  x;
  
  const int DIM = 1 << DIM2;

  prefix[tid] = value;
  __syncthreads();

#if 1
  x = prefix[tid -  1]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  1); __syncthreads();
  x = prefix[tid -  2]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  2); __syncthreads();
  x = prefix[tid -  4]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  4); __syncthreads();
  x = prefix[tid -  8]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  8); __syncthreads();
  x = prefix[tid - 16]; __syncthreads(); prefix[tid] += x & BTEST(tid >= 16); __syncthreads();
  if (DIM2 >= 6) {x = prefix[tid - 32]; __syncthreads(); prefix[tid] += x & BTEST(tid >= 32); __syncthreads();}
  if (DIM2 >= 7) {x = prefix[tid - 64]; __syncthreads(); prefix[tid] += x & BTEST(tid >= 64); __syncthreads();}
  if (DIM2 >= 8) {x = prefix[tid -128]; __syncthreads(); prefix[tid] += x & BTEST(tid >=128); __syncthreads();}

  x = prefix[DIM - 1];
  __syncthreads();
  return x;
#else
  
  int offset = 0;
  int tid2 = tid << 1;

#pragma unroll
  for (int d = DIM >> 1; d > 0; d >>= 1) {
    __syncthreads();

    int iflag = BTEST(tid < d);
    int ai = (((tid2 + 1) << offset) - 1) & iflag;
    int bi = (((tid2 + 2) << offset) - 1) & iflag;
    
    prefix[bi] += prefix[ai] & iflag;
    offset++;
  }

  // clear the last element
  if (tid == 0) prefix[DIM - 1] = 0;

  // traverse down the tree building the scan in place
#pragma unroll
  for (int d = 1; d < DIM; d <<= 1) {
    offset--;
    __syncthreads();
    
    int iflag = BTEST(tid < d);
    int ai = (((tid2 + 1) << offset) - 1) & iflag;
    int bi = (((tid2 + 2) << offset) - 1) & iflag;
    
    int t       = prefix[ai];
    if (tid < d) {
      prefix[ai]  = (prefix[bi] & iflag) + (t & BTEST(tid >= d));
      prefix[bi] += t & iflag;
    }
  }
  __syncthreads();

  prefix[tid] += value;
  __syncthreads();
  
  x = prefix[DIM - 1];
  __syncthreads();
  return x;
#endif
}

template<int SHIFT>
__forceinline__ __device__ int ACCS(const int i)
{
  return (i & ((LMEM_STACK_SIZE << SHIFT) - 1))*blockDim.x + threadIdx.x;
}


#define BTEST(x) (-(int)(x))

texture<float4, 1, hipReadModeElementType> texNodeSize;
texture<float4, 1, hipReadModeElementType> texNodeCenter;
texture<float4, 1, hipReadModeElementType> texMultipole;
texture<float4, 1, hipReadModeElementType> texBody;

template<class T>
 struct ADDOP {
  __device__ static inline T identity()           {return (T)(0);}
  __device__ static inline T apply(T a, T b)      {return (T)(a + b);};
  __device__ static inline T unapply(T a, T b)    {return (T)(a - b);};
  __device__ static inline T mask(bool flag, T b) {return (T)(-(int)(flag) & b);};
};

template<class OP, class T>
// __device__ T inclusive_scan_warp(volatile T *ptr, T mysum,  const unsigned int idx = threadIdx.x) {
__device__ __forceinline__ T inclusive_scan_warp(volatile T *ptr, T mysum,  const unsigned int idx ) {
  const unsigned int lane = idx & 31;

  if (lane >=  1) ptr[idx] = mysum = OP::apply(ptr[idx -  1], mysum);
  if (lane >=  2) ptr[idx] = mysum = OP::apply(ptr[idx -  2], mysum);
  if (lane >=  4) ptr[idx] = mysum = OP::apply(ptr[idx -  4], mysum);
  if (lane >=  8) ptr[idx] = mysum = OP::apply(ptr[idx -  8], mysum);
  if (lane >= 16) ptr[idx] = mysum = OP::apply(ptr[idx - 16], mysum);

  return ptr[idx];
}


__device__ __forceinline__ int inclusive_scan_warp(volatile int *ptr, int mysum, const unsigned int idx) {

  const unsigned int lane = idx & 31;

  if (lane >=  1) ptr[idx] = mysum = ptr[idx -  1]   + mysum;
  if (lane >=  2) ptr[idx] = mysum = ptr[idx -  2]   + mysum;
  if (lane >=  4) ptr[idx] = mysum = ptr[idx -  4]   + mysum;
  if (lane >=  8) ptr[idx] = mysum = ptr[idx -  8]   + mysum;
  if (lane >= 16) ptr[idx] = mysum = ptr[idx -  16]  + mysum;

  return ptr[idx];
}


template<class OP, class T>
__device__ __inline__ T inclusive_scan_block(volatile T *ptr, const T v0, const unsigned int idx) {
  const unsigned int lane   = idx & 31;
  const unsigned int warpid = idx >> 5;

  // step 0: Write the valume from the thread to the memory
  ptr[idx] = v0;
  T mysum = v0;
  __syncthreads();

  // step 1: Intra-warp scan in each warp
//   T val = inclusive_scan_warp<OP, T>(ptr, mysum, idx);
  T val = inclusive_scan_warp(ptr, mysum, idx);
  __syncthreads();

  // step 2: Collect per-warp particle results
  if (lane == 31) ptr[warpid] =  ptr[idx];
  __syncthreads();

  mysum =  ptr[idx];

  // step 3: Use 1st warp to scan per-warp results
  if (warpid == 0) inclusive_scan_warp<OP, T>(ptr,mysum, idx);
  __syncthreads();

  // step 4: Accumulate results from Steps 1 and 3;
  if (warpid > 0) val = OP::apply(ptr[warpid - 1], val);
  __syncthreads();

  // Step 5: Write and return the final result
  ptr[idx] = val;
  __syncthreads();

  return val; //ptr[blockDim.x - 1];
}



template<class OP, class T>
// __device__ T inclusive_scan_block(volatile T *ptr, const unsigned int idx = threadIdx.x) {
__device__ T inclusive_scan_block(volatile T *ptr, const unsigned int idx) {
  const unsigned int lane   = idx & 31;
  const unsigned int warpid = idx >> 5;

   T mysum = ptr[idx];
   __syncthreads();

  // step 1: Intra-warp scan in each warp
  T val = inclusive_scan_warp<OP, T>(ptr, mysum, idx);
  __syncthreads();

  // step 2: Collect per-warp particle results
  if (lane == 31) ptr[warpid] = ptr[idx];
  __syncthreads();

  mysum = ptr[idx];

  // step 3: Use 1st warp to scan per-warp results
  if (warpid == 0) inclusive_scan_warp<OP, T>(ptr,mysum, idx);
  __syncthreads();

  // step 4: Accumulate results from Steps 1 and 3;
  if (warpid > 0) val = OP::apply(ptr[warpid - 1], val);
  __syncthreads();

  // Step 5: Write and return the final result
  ptr[idx] = val;
  __syncthreads();

  return val; //ptr[blockDim.x - 1];
}


template<class OP, class T>
// __device__ T inclusive_scan_array(volatile T *ptr_global, const int N, const unsigned int idx = threadIdx.x) {
__device__ T inclusive_scan_array(volatile T *ptr_global, const int N, const unsigned int idx) {


  T y = OP::identity();
  volatile T *ptr = ptr_global;

  for (int p = 0; p < N; p += blockDim.x) {
    ptr = &ptr_global[p];
    inclusive_scan_block<OP, T>(ptr, idx);
    ptr[idx] = OP::apply(ptr[idx], y);
    __syncthreads();

    y = ptr[blockDim.x - 1];
    __syncthreads();
  }

  return y;

}

/*********** Forces *************/

__device__ float4 add_acc(
        float4 acc,  const float4 pos,
			  const float massj, const float3 posj,
			  const float eps2)
{
#if 1  /* to test performance of a tree-walk */
  const float3 dr = make_float3(posj.x - pos.x, posj.y - pos.y, posj.z - pos.z);

  const float r2     = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z + eps2;
  const float rinv   = rsqrtf(r2);
  const float rinv2  = rinv*rinv;
  const float mrinv  = massj * rinv;
  const float mrinv3 = mrinv * rinv2;

  acc.w -= mrinv;
  acc.x += mrinv3 * dr.x;
  acc.y += mrinv3 * dr.y;
  acc.z += mrinv3 * dr.z;
#endif

  return acc;
}


//Improved Barnes Hut criterium
__device__ bool split_node_grav_impbh(
    const float4 nodeCOM, 
    const float4 groupCenter, 
    const float4 groupSize)
{
  //Compute the distance between the group and the cell
  float3 dr = make_float3(
      fabsf(groupCenter.x - nodeCOM.x) - (groupSize.x),
      fabsf(groupCenter.y - nodeCOM.y) - (groupSize.y),
      fabsf(groupCenter.z - nodeCOM.z) - (groupSize.z)
      );

  dr.x += fabsf(dr.x); dr.x *= 0.5f;
  dr.y += fabsf(dr.y); dr.y *= 0.5f;
  dr.z += fabsf(dr.z); dr.z *= 0.5f;

  //Distance squared, no need to do sqrt since opening criteria has been squared
  const float ds2    = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;

  return (ds2 <= fabsf(nodeCOM.w));
}



#define TEXTURES
#define OLDPREFIX
#if 0
#define _ORIG_SHMEM_
#endif


template<int DIM2, int SHIFT>
__device__ float4 approximate_gravity(int DIM2x, int DIM2y,
    int tid, int tx, int ty,
    int body_i, float4 pos_i,
    real4 group_pos,
    float eps2,
    uint2 node_begend,
    real4 *multipole_data,
    real4 *body_pos,
    int *shmem,
    int *lmem,
    int &ngb,
    int &apprCount, int &direCount,
    volatile float4 *boxSizeInfo,
    float4 groupSize,
    volatile float4 *boxCenterInfo,
    float group_eps,
    real4 *body_vel) {

  float4 acc_i = {0.0f, 0.0f, 0.0f, 0.0f};


  /*********** set necessary thread constants **********/

  const int DIMx = 1  << DIM2x;
  const int DIMy = 1  << DIM2y;
  const int DIM  = 1  << DIM2;
  const int offs = ty << DIM2x;

  /*********** shared memory distribution **********/

  //  begin,    end,   size
  // -----------------------
#ifdef _ORIG_SHMEM_
  
  int *approxS = (int*)&shmem  [     0];            //  0*DIM,  2*DIM,  2*DIM
  int *directS = (int*)&approxS[ 2*DIM];            //  2*DIM,  3*DIM,  1*DIM
  int *nodesS = (int*)&directS [   DIM];            //  3*DIM, 12*DIM,  9*DIM
  int *prefix = (int*)&nodesS  [9 *DIM];            // 12*DIM, 14*DIM,  2*DIM
  int *sh_body = &approxS[DIM];
  
  int *prefix0 = &prefix[  0];
  int *prefix1 = &prefix[DIM];
  
  const int NJMAX = DIM*2;
  int    *body_list = (int*   )&nodesS   [  DIM]; //  4*DIM,  6*DIM,  2*DIM
  float  *sh_mass   = (float* )&body_list[NJMAX]; //  6*DIM,  7*DIM,  1*DIM
  float3 *sh_pos    = (float3*)&sh_mass  [  DIM]; //  7*DIM, 10*DIM   3*DIM
  
  int *approxM = approxS;
  int *directM = directS;
  int * nodesM =  nodesS;

#else   /* !_ORIG_SHMEM_ */

  const int stack_sz = (LMEM_STACK_SIZE << SHIFT) << DIM2;  /* stack allocated per thread-block */
  int *approxL = lmem + stack_sz; 

  int *directS = shmem;                              //  0*DIM,  1*DIM,  1*DIM
  int *nodesS  = directS + DIM;                      //  1*DIM, 10*DIM,  9*DIM
  int *prefix  = nodesS  + DIM*9;                    // 10*DIM, 12*DIM,  2*DIM
  
  int *prefix0 = &prefix[  0];
  int *prefix1 = &prefix[DIM];
  
  const int NJMAX = DIM*3;
  int    *body_list = (int*   )&nodesS   [  DIM]; //  2*DIM,   5*DIM,  2*DIM
  float  *sh_mass   = (float* )&body_list[NJMAX]; //  5*DIM,   6*DIM,  1*DIM
  float3 *sh_pos    = (float3*)&sh_mass  [  DIM]; //  6*DIM,   9*DIM   3*DIM
  int    *sh_body   = nodesS + DIM*8;             //  9*DIM,  10*DIM,  1*DIM
  
  int *approxM = approxL;
  int *directM = directS;
  int * nodesM =  nodesS;

#endif /* _ORIG_SHMEM_ */


  float  *node_mon0 = sh_mass;
  float3 *node_mon1 = sh_pos; 
  
  float  *sh_pot = sh_mass;
  float3 *sh_acc = sh_pos;

  /*********** stack **********/

  int *nstack = lmem;

  /*********** begin tree-walk **********/

  int n_approx = 0;
  int n_direct = 0;


  for (int root_node = node_begend.x; root_node < node_begend.y; root_node += DIM) 
  {
    int n_nodes0 = min(node_begend.y - root_node, DIM);
    int n_stack0 = 0;
    int n_stack_pre = 0;

    { nstack[ACCS<SHIFT>(n_stack0)] = root_node + tid;   n_stack0++; }

    /*********** walk each level **********/
    while (n_nodes0 > 0) {


      int n_nodes1 = 0;
      int n_offset = 0;

      int n_stack1 = n_stack0;
      int c_stack0 = n_stack_pre;

      /*********** walk a level **********/
      while(c_stack0 < n_stack0) 
      {

        /***
         **** --> fetch the list of nodes rom LMEM
         ***/
        bool use_node = tid <  n_nodes0;
#if 0
        { prefix[tid] = nstack[ACCS<SHIFT>(c_stack0)];   c_stack0++; }
        __syncthreads();
        int node  = prefix[min(tid, n_nodes0 - 1)];
#else  /* eg: seems to work, but I do not remember if that will *always* work */
        int node;
        { node  = nstack[ACCS<SHIFT>(c_stack0)];   c_stack0++; }
#endif

#if 0
        if(n_nodes0 > 0){       //Work around pre 4.1 compiler bug
          n_nodes0 -= DIM;
        }
#else
        n_nodes0 -= DIM;
#endif

        /***
         **** --> process each of the nodes in the list in parallel
         ***/

#ifndef TEXTURES
        float4 nodeSize = boxSizeInfo[node];                   //Fetch the size of the box. Size.w = child info
        float4 node_pos = boxCenterInfo[node];                 //Fetch the center of the box. center.w = opening info
#else
        float4 nodeSize =  tex1Dfetch(texNodeSize, node);
        float4 node_pos =  tex1Dfetch(texNodeCenter, node);
#endif

        int node_data = __float_as_int(nodeSize.w);

        //Check if a cell has to be opened
#ifndef TEXTURES
        float4 nodeCOM = multipole_data[node*3];
#else
        float4 nodeCOM = tex1Dfetch(texMultipole,node*3);
#endif

        nodeCOM.w      = node_pos.w;
        bool   split   = split_node_grav_impbh(nodeCOM, group_pos, groupSize);


        bool leaf       = node_pos.w <= 0;  //Small AND equal incase of a 1 particle cell       //Check if it is a leaf
        //         split = true;

        uint mask    = BTEST((split && !leaf) && use_node);               // mask = #FFFFFFFF if use_node+split+not_a_leaf==true, otherwise zero
        int child    =    node_data & 0x0FFFFFFF;                         //Index to the first child of the node
        int nchild   = (((node_data & 0xF0000000) >> 28)) & mask;         //The number of children this node has

        /***
         **** --> calculate prefix
         ***/


#ifdef OLDPREFIX
        int n_total = calc_prefix<DIM2>(prefix, tid,  nchild);
        prefix[tid] += n_offset - nchild;
        __syncthreads();
#else
        inclusive_scan_block<ADDOP<int>, int>(prefix, nchild, tid);        // inclusive scan to compute memory offset of each child
        int n_total = prefix[blockDim.x - 1];                              // fetch total number of children, i.e. offset of the last child -1
        __syncthreads();                                                   // thread barrier to make sure that warps completed their jobs
        prefix[tid] += n_offset - nchild;                                  // convert inclusive into exclusive scan for referencing purpose
        __syncthreads();                                                   // thread barrier
#endif

        for (int i = n_offset; i < n_offset + n_total; i += DIM)         //nullify part of the array that will be filled with children
          nodesM[tid + i] = 0;                                          //but do not touch those parts which has already been filled
        __syncthreads();                                                 //Thread barrier to make sure all warps finished writing data

        bool flag = (split && !leaf) && use_node;                        //Flag = use_node + split + not_a_leaf;Use only non_leaf nodes that are to be split
#if 1
        if (flag) nodesM[prefix[tid]] = child;                            //Thread with the node that is about to be split
        __syncthreads();                                                 //writes the first child in the array of nodes

        /*** in the following 8 lines, we calculate indexes of all the children that have to be walked from the index of the first child***/
        if (flag && nodesM[prefix[tid] + 1] == 0) nodesM[prefix[tid] + 1] = child + 1; __syncthreads();
        if (flag && nodesM[prefix[tid] + 2] == 0) nodesM[prefix[tid] + 2] = child + 2; __syncthreads();
        if (flag && nodesM[prefix[tid] + 3] == 0) nodesM[prefix[tid] + 3] = child + 3; __syncthreads();
        if (flag && nodesM[prefix[tid] + 4] == 0) nodesM[prefix[tid] + 4] = child + 4; __syncthreads();
        if (flag && nodesM[prefix[tid] + 5] == 0) nodesM[prefix[tid] + 5] = child + 5; __syncthreads();
        if (flag && nodesM[prefix[tid] + 6] == 0) nodesM[prefix[tid] + 6] = child + 6; __syncthreads();
        if (flag && nodesM[prefix[tid] + 7] == 0) nodesM[prefix[tid] + 7] = child + 7; __syncthreads();
#else
#if 1
        if (flag) nodesM[prefix[tid]] = child;                            //Thread with the node that is about to be split
                                                                          //writes the first child in the array of nodes
#else
        const int maskT = flag ? 0xFFFFFFFF : 0x0;
        const int maskF = ~maskT;
        const int addr = (prefix[tid] & maskT) + (-1 & maskF);;
        nodesM[addr] = (maskF & nodesM[addr]) + (maskT & child);
#endif
        
        if (flag && nodesM[prefix[tid] + 1] == 0) nodesM[prefix[tid] + 1] = child + 1; 
        if (flag && nodesM[prefix[tid] + 2] == 0) nodesM[prefix[tid] + 2] = child + 2;
        if (flag && nodesM[prefix[tid] + 3] == 0) nodesM[prefix[tid] + 3] = child + 3;
        if (flag && nodesM[prefix[tid] + 4] == 0) nodesM[prefix[tid] + 4] = child + 4;
        if (flag && nodesM[prefix[tid] + 5] == 0) nodesM[prefix[tid] + 5] = child + 5;
        if (flag && nodesM[prefix[tid] + 6] == 0) nodesM[prefix[tid] + 6] = child + 6;
        if (flag && nodesM[prefix[tid] + 7] == 0) nodesM[prefix[tid] + 7] = child + 7;
        __syncthreads();
#endif

        n_offset += n_total;    //Increase the offset in the array by the number of newly added nodes


        /***
         **** --> save list of nodes to LMEM
         ***/

        /*** if half of shared memory or more is filled with the the nodes, dump these into slowmem stack ***/
        while(n_offset >= DIM) 
        {
          n_offset -= DIM;
          const int offs1 = ACCS<SHIFT>(n_stack1);
          nstack[offs1] = nodesM[n_offset + tid];   n_stack1++;
          n_nodes1 += DIM;

          if((n_stack1 - c_stack0) >= (LMEM_STACK_SIZE << SHIFT))
          {
            //We overwrote our current stack
            apprCount = -1; 
            return acc_i;	 
          }
        }

        __syncthreads();



        /******************************/
        /******************************/
        /*****     EVALUATION     *****/
        /******************************/
        /******************************/
#if 1
        /***********************************/
        /******       APPROX          ******/
        /***********************************/

#ifdef OLDPREFIX
        n_total = calc_prefix<DIM2>(prefix, tid,  1 - (split || !use_node));
#else
        inclusive_scan_block<ADDOP<int>, int>(prefix, 1 - (split || !use_node), tid);
        n_total = prefix[blockDim.x - 1];
#endif


        // 	n_total = calc_prefix<DIM2>(prefix, tid,  !split && use_node);         // for some unkown reason this does not work right on the GPU
        if (!split && use_node) approxM[n_approx + prefix[tid] - 1] = node;
        __syncthreads();
        n_approx += n_total;

        while (n_approx >= DIM) 
        {
          n_approx -= DIM;
          const int address      = (approxM[n_approx + tid] << 1) + approxM[n_approx + tid];
#ifndef TEXTURES
          const float4 monopole  = multipole_data[address    ];
#if 0
          float4 octopole0 = multipole_data[address + 1];
          float4 octopole1 = multipole_data[address + 2];
#endif
#else
          const float4 monopole  = tex1Dfetch(texMultipole, address);
#if 0
          float4 octopole0 = tex1Dfetch(texMultipole, address + 1);
          float4 octopole1 = tex1Dfetch(texMultipole, address + 2);
#endif
#endif

          node_mon0[tid] = monopole.w;
          node_mon1[tid] = make_float3(monopole.x,  monopole.y,  monopole.z);
          __syncthreads();

#if 0
          const float f_dm   = 0.0f;
          const float f_star = 1.0f
            const float darkMatterMass = f_dm   * octopole1.w;
          /* eg: we need to be careful with the line below to avoid truncation error due to 
             subtraction of two large numbers, monopole.w and darkMatterMass both could be
             very large.
             Instead, we can use octopole1.w to be stellar mass, and DM mass to be 
             monopole.w, then we add the two together to get total mass, but this will
             require more changes to the kernel */
          const float    stellarMass = f_star * (monopole.w - darkMatterMass);
          const float hinv = 1.0f/hi;   /* eg: this can be precomputing to avoid division */
          density += interact(
              make_float3(pos_i.x, pos_i.y, pos_i.z), h, hinv,
              make_float3(monopole.x, monople.y, monopole.z), darkMatterMass + stellarMass);
          /* eg: the interact function still calls sqrtf(f), which invloves 1 div and 1 rsqrtf,
             so ideally we would like to take advantage of rsqrtf in add_acc, and then we only
             do 1 div */
#endif


#if 1
#pragma unroll 16
          for (int i = 0; i < DIMx; i++)
            acc_i = add_acc(acc_i, pos_i, node_mon0[offs + i], node_mon1[offs+i], eps2);
          apprCount += DIMx;
          __syncthreads();
#endif
        }
        __syncthreads();
#endif

#if 1
        /***********************************/
        /******       DIRECT          ******/
        /***********************************/


        flag         = split && leaf && use_node;                                //flag = split + leaf + use_node
        int  jbody   = node_data & BODYMASK;                                     //the first body in the leaf
        int  nbody   = (((node_data & INVBMASK) >> LEAFBIT)+1) & BTEST(flag);    //number of bodies in the leaf masked with the flag

        body_list[tid] = directM[tid];                                            //copy list of bodies from previous pass to body_list
        sh_body  [tid] = jbody;                                                  //store the leafs first body id into shared memory

        // step 1
#ifdef OLDPREFIX
        calc_prefix<DIM2>(prefix0, tid, flag);
#else
        inclusive_scan_block<ADDOP<int>, int>(prefix0, (int)flag, tid);       // inclusive scan on flags to construct array
#endif

        if (flag) prefix1[prefix0[tid] - 1] = tid;                             //with tidś whose leaves have to be opened
        __syncthreads();                                                      //thread barrier, make sure all warps completed the job

        // step 2
#ifdef OLDPREFIX
        int n_bodies  = calc_prefix<DIM2>(prefix0, tid, nbody);
#else
        inclusive_scan_block<ADDOP<int>, int>(prefix0, nbody, tid);        // inclusive scan to compute memory offset for each body
        int n_bodies = prefix0[blockDim.x - 1];                            //Total number of bides extract from the leaves
        __syncthreads();                                                   // thread barrier to make sure that warps completed their jobs
#endif

        directM[tid]  = prefix0[tid];                                       //Store a copy of inclusive scan in direct
        prefix0[tid] -= nbody;                                              //convert inclusive int oexclusive scan
        prefix0[tid] += 1;                                                  //add unity, since later prefix0[tid] == 0 used to check barrier

        int nl_pre = 0;                                                     //Number of leaves that have already been processed

        while (n_bodies > 0) 
        {
          int nb    = min(n_bodies, NJMAX - n_direct);                    //Make sure number of bides to be extracted does not exceed
          //the amount of allocated shared memory

          // step 0                                                      //nullify part of the body_list that will be filled with bodies
          for (int i = n_direct; i < n_direct + nb; i += DIM){           //from the leaves that are being processed
            body_list[i + tid] = 0;
          }
          __syncthreads();

          //step 1:
          if (flag && (directM[tid] <= nb) && (prefix0[tid] > 0))        //make sure that the thread indeed carries a leaf
            body_list[n_direct + prefix0[tid] - 1] = 1;                 //whose bodies will be extracted
          __syncthreads();

          //step 2:
#ifdef OLDPREFIX
          int nl = calc_prefix<DIM2>(nb, &body_list[n_direct], tid);
#else
          int nl = inclusive_scan_array<ADDOP<int>, int>              // inclusive scan to compute number of leaves to process
            (&body_list[n_direct], nb, tid);            // to make sure that there is enough shared memory for bodies
#endif
          nb = directM[prefix1[nl_pre + nl - 1]];                        // number of bodies stored in these leaves

          // step 3:
          for (int i = n_direct; i < n_direct + nb; i += DIM) {          //segmented fill of the body_list
            int j = prefix1[nl_pre + body_list[i + tid] - 1];            // compute the first body in shared j-body array
            body_list[i + tid] = (i + tid - n_direct) -                 //add to the index of the first j-body in a child
              (prefix0[j] - 1) + sh_body[j];         //the index of the first child in body_list array
          }
          __syncthreads();


          /**************************************************
           *  example of what is accomplished in steps 0-4   *
           *       ---------------------------               *
           * step 0: body_list = 000000000000000000000       *
           * step 1: body_list = 100010001000000100100       *
           * step 2: body_list = 111122223333333444555       *
           * step 3: body_list = 012301230123456012012       *
           *         assuming that sh_body[j] = 0            *
           ***************************************************/

          n_bodies     -= nb;                                   //subtract from n_bodies number of bodies that have been extracted
          nl_pre       += nl;                                   //increase the number of leaves that where processed
          directM[tid] -= nb;                                   //subtract the number of extracted bodies in this pass
          prefix0[tid] = max(prefix0[tid] - nb, 0);             //same here, but do not let the number be negative (GT200 bug!?)
          n_direct     += nb;                                  //increase the number of bodies to be procssed

          while(n_direct >= DIM) 
          {
            n_direct -= DIM;


            const float4 posj  = body_pos[body_list[n_direct + tid]];
#if 0
            const float4 posj  = tex1Dfetch(texBody, body_list[n_direct + tid]);
#endif
            sh_mass[tid] = posj.w;
            sh_pos [tid] = make_float3(posj.x, posj.y, posj.z);

            __syncthreads();
#if 1
#pragma unroll 16
            for (int j = 0; j < DIMx; j++)
              acc_i = add_acc(acc_i, pos_i, sh_mass[offs + j], sh_pos[offs + j], eps2);
            direCount += DIMx;
            __syncthreads();
#endif
          }

        }
        directM[tid] = body_list[tid];
        __syncthreads();
#endif
      } //end lvl


      n_nodes1 += n_offset;
      if (n_offset > 0)
      { 
        nstack[ACCS<SHIFT>(n_stack1)] = nodesM[tid];   n_stack1++; 
        if((n_stack1 - c_stack0) >= (LMEM_STACK_SIZE << SHIFT))
        {
          //We overwrote our current stack
          apprCount = -1; 
          return acc_i;	 
        }
      }
      __syncthreads();


      /***
       **** --> copy nodes1 to nodes0: done by reassigning the pointers
       ***/
      n_nodes0    = n_nodes1;

      n_stack_pre = n_stack0;
      n_stack0    = n_stack1;

    }//end while   levels
  }//end for


  if(n_approx > 0)
  {
    if (tid < n_approx) 
    {
      const int address = (approxM[tid] << 1) + approxM[tid];
#ifndef TEXTURES
      float4 monopole  = multipole_data[address    ];
      float4 octopole0 = multipole_data[address + 1];
      float4 octopole1 = multipole_data[address + 2];
#else
      float4 monopole  = tex1Dfetch(texMultipole, address);
      float4 octopole0 = tex1Dfetch(texMultipole, address + 1);
      float4 octopole1 = tex1Dfetch(texMultipole, address + 2);
#endif

      node_mon0[tid] = monopole.w;
      node_mon1[tid] = make_float3(monopole.x,  monopole.y,  monopole.z);

    } else {

      //Set non-active memory locations to zero
      node_mon0[tid] = 0.0f;
      node_mon1[tid] = make_float3(1.0e10f, 1.0e10f, 1.0e10f);

    }
    __syncthreads();
#pragma unroll
    for (int i = 0; i < DIMx; i++)
      acc_i = add_acc(acc_i, pos_i, node_mon0[offs + i], node_mon1[offs+i],eps2);
    apprCount += DIMx;

    __syncthreads();
  } //if n_approx > 0

  if(n_direct > 0)
  {
    if (tid < n_direct) 
    {
      const float4 posj = body_pos[directM[tid]];
#if 0
      const float4 posj  = tex1Dfetch(texBody, direct[tid]);
#endif
      sh_mass[tid] = posj.w;
      sh_pos [tid] = make_float3(posj.x, posj.y, posj.z);
    } else {
      sh_mass[tid] = 0.0f;
      sh_pos [tid] = make_float3(1.0e10f, 1.0e10f, 1.0e10f);
    }

    __syncthreads();
#pragma unroll
    for (int j = 0; j < DIMx; j++) 
      acc_i = add_acc(acc_i, pos_i, sh_mass[offs + j], sh_pos[offs + j], eps2);
    direCount += DIMx;
    __syncthreads();
  }

  /***
   **** --> reduce data between threads
   ***/
  sh_pot[tid] = acc_i.w;
  sh_acc[tid] = make_float3(acc_i.x, acc_i.y, acc_i.z);
  __syncthreads();

  if (ty == 0) 
#pragma unroll
    for (int i = 1; i < DIMy; i++) 
    {
      const int idx = (i << DIM2x) + tx;
      acc_i.w += sh_pot[idx];
      acc_i.x += sh_acc[idx].x;
      acc_i.y += sh_acc[idx].y;
      acc_i.z += sh_acc[idx].z;
    }
  __syncthreads();


  //Sum the interaction counters
  float  *sh_ds2 = (float*)&sh_acc[DIM];
  int    *sh_ngb = (int*  )&sh_ds2[DIM];
  sh_ds2[tid] = direCount;
  sh_ngb[tid] = apprCount;

  __syncthreads();


  if (ty == 0) {
#pragma unroll
    for (int i = 1; i < DIMy; i++){
      int idx = (i << DIM2x) + tx;
      direCount  += sh_ds2[idx];
      apprCount  += sh_ngb[idx];
    }
  }
  __syncthreads();

  return acc_i;
}


  extern "C" __global__ void
__launch_bounds__(NTHREAD)
  dev_approximate_gravity(const int n_active_groups,
      int    n_bodies,
      float eps2,
      uint2 node_begend,
      int    *active_groups,
      real4  *body_pos,
      real4  *multipole_data,
      float4 *acc_out,
      real4  *group_body_pos,
      int    *ngb_out,
      int    *active_inout,
      int2   *interactions,
      float4  *boxSizeInfo,
      float4  *groupSizeInfo,
      float4  *boxCenterInfo,
      float4  *groupCenterInfo,
      real4   *body_vel,
      int     *MEM_BUF) {
    //                                                    int     grpOffset){


    const int blockDim2 = NTHREAD2;
#ifdef _ORIG_SHMEM_
    __shared__ int shmem[15*(1 << blockDim2)];
#else
    __shared__ int shmem[12*(1 << blockDim2)];
#endif
    //    __shared__ int shmem[24*(1 << blockDim2)]; is possible on FERMI
    //    int             lmem[LMEM_STACK_SIZE];



    /*********** check if this block is linked to a leaf **********/

    int bid = gridDim.x * blockIdx.y + blockIdx.x;

    while(true)
    {

      if(threadIdx.x == 0)
      {
        bid         = atomicAdd(&active_inout[n_bodies], 1);
        shmem[0]    = bid;
      }
      __syncthreads();

      bid   = shmem[0];

      if (bid >= n_active_groups) return;


      int tid = threadIdx.y * blockDim.x + threadIdx.x;

      int grpOffset = 0;

      //   volatile int *lmem = &MEM_BUF[blockIdx.x*LMEM_STACK_SIZE*blockDim.x + threadIdx.x*LMEM_STACK_SIZE];
      //   int *lmem = &MEM_BUF[blockIdx.x*LMEM_STACK_SIZE*blockDim.x + threadIdx.x*LMEM_STACK_SIZE];
#ifdef _ORIG_SHMEM_
      int *lmem = &MEM_BUF[blockIdx.x* LMEM_STACK_SIZE*blockDim.x];
#else
      int *lmem = &MEM_BUF[blockIdx.x*(LMEM_STACK_SIZE*blockDim.x + LMEM_EXTRA_SIZE)];
#endif


      /*********** set necessary thread constants **********/
#ifdef DO_BLOCK_TIMESTEP
      real4 curGroupSize    = groupSizeInfo[active_groups[bid + grpOffset]];
#else
      real4 curGroupSize    = groupSizeInfo[bid + grpOffset];
#endif
      int   groupData       = __float_as_int(curGroupSize.w);
      uint body_i           =   groupData & CRITMASK;
      uint nb_i             = ((groupData & INVCMASK) >> CRITBIT) + 1;

#ifdef DO_BLOCK_TIMESTEP
      real4 group_pos       = groupCenterInfo[active_groups[bid + grpOffset]];
#else
      real4 group_pos       = groupCenterInfo[bid + grpOffset];
#endif
      //   if(tid == 0)
      //   printf("[%f %f %f %f ] \n [%f %f %f %f ] %d %d \n",
      //           curGroupSize.x, curGroupSize.y, curGroupSize.z, curGroupSize.w,
      //           group_pos.x, group_pos.y, group_pos.z, group_pos.w, body_i, nb_i);


      int DIM2x = 0;
      while (((nb_i - 1) >> DIM2x) > 0) DIM2x++;

      DIM2x     = max(DIM2x,4);
      int DIM2y = blockDim2 - DIM2x;

      int tx = tid & ((1 << DIM2x) - 1);
      int ty = tid >> DIM2x;

      body_i += tx%nb_i;

      //float4 pos_i = tex1Dfetch(bodies_pos_ref, body_i);   // texture read: 4 floats


//       float4 pos_i = body_pos[body_i];
      float4 pos_i = group_body_pos[body_i];



      int ngb_i;

      float4 acc_i = {0.0f, 0.0f, 0.0f, 0.0f};

#ifdef INDSOFT
      eps2 = body_vel[body_i].w;
      float group_eps = eps2;

      volatile float *reduc = (float*) &shmem[0];
      reduc[threadIdx.x] = eps2;

      //Find the maximum softening value for the particles in this group
      __syncthreads();
      // do reduction in shared mem
      if(blockDim.x >= 512) if (tid < 256) {reduc[threadIdx.x] = group_eps = fmaxf(group_eps, reduc[threadIdx.x + 256]);} __syncthreads();
      if(blockDim.x >= 256) if (tid < 128) {reduc[threadIdx.x] = group_eps = fmaxf(group_eps, reduc[threadIdx.x + 128]);} __syncthreads();
      if(blockDim.x >= 128) if (tid < 64)  {reduc[threadIdx.x] = group_eps = fmaxf(group_eps, reduc[threadIdx.x + 64]);} __syncthreads();
      if(blockDim.x >= 64) if (tid < 32) { reduc[threadIdx.x] = group_eps = fmaxf(group_eps, reduc[threadIdx.x + 32]);}
      if(blockDim.x >= 32) if (tid < 16) { reduc[threadIdx.x] = group_eps = fmaxf(group_eps, reduc[threadIdx.x + 16]);}

      if(tid < 8)
      {
        reduc[threadIdx.x] = group_eps = fmaxf(group_eps, reduc[threadIdx.x + 8]);
        reduc[threadIdx.x] = group_eps = fmaxf(group_eps, reduc[threadIdx.x + 4]);
        reduc[threadIdx.x] = group_eps = fmaxf(group_eps, reduc[threadIdx.x + 2]);
        reduc[threadIdx.x] = group_eps = fmaxf(group_eps, reduc[threadIdx.x + 1]);
      }
      __syncthreads();

      group_eps = reduc[0];
#else
      float group_eps  = 0;
#endif

      int apprCount = 0;
      int direCount = 0;


      acc_i = approximate_gravity<blockDim2, 0>( DIM2x, DIM2y, tid, tx, ty,
          body_i, pos_i, group_pos,
          eps2, node_begend,
          multipole_data, body_pos,
          shmem, lmem, ngb_i, apprCount, direCount, boxSizeInfo, curGroupSize, boxCenterInfo,
          group_eps, body_vel);
      if(apprCount < 0)
      {

        //Try to get access to the big stack, only one block per time is allowed
        if(threadIdx.x == 0)
        {
          int res = atomicExch(&active_inout[n_bodies+1], 1); //If the old value (res) is 0 we can go otherwise sleep
          int waitCounter  = 0;
          while(res != 0)
          {
            //Sleep
            for(int i=0; i < (1024); i++)
            {
              waitCounter += 1;
            }
            //Test again
            shmem[0] = waitCounter;
            res = atomicExch(&active_inout[n_bodies+1], 1); 
          }
        }

        __syncthreads();

#ifdef _ORIG_SHMEM_
        lmem = &MEM_BUF[gridDim.x*LMEM_STACK_SIZE*blockDim.x];    //Use the extra large buffer
#else
        lmem = &MEM_BUF[gridDim.x*(LMEM_STACK_SIZE*blockDim.x + LMEM_EXTRA_SIZE)];    //Use the extra large buffer
#endif
        apprCount = direCount = 0;
        acc_i = approximate_gravity<blockDim2, 8>( DIM2x, DIM2y, tid, tx, ty,
            body_i, pos_i, group_pos,
            eps2, node_begend,
            multipole_data, body_pos,
            shmem, lmem, ngb_i, apprCount, direCount, boxSizeInfo, curGroupSize, boxCenterInfo,
            group_eps, body_vel);

#ifdef _ORIG_SHMEM_
        lmem = &MEM_BUF[blockIdx.x* LMEM_STACK_SIZE*blockDim.x]; //Back to normal location
#else
        lmem = &MEM_BUF[blockIdx.x*(LMEM_STACK_SIZE*blockDim.x + LMEM_EXTRA_SIZE)];
#endif

        if(threadIdx.x == 0)
        {
          atomicExch(&active_inout[n_bodies+1], 0); //Release the lock
        }
      }//end if apprCount < 0

      if (tid < nb_i) {
        acc_out     [body_i] = acc_i;
        ngb_out     [body_i] = -1;
        active_inout[body_i] = 1;
        interactions[body_i].x = apprCount;
        interactions[body_i].y = direCount ;
      }


    }     //end while
  }
